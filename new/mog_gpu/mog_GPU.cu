
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void mog_malloc_gpu(int64_t device_id, 
				int64_t slab_size, 
				char** slabs,
				int64_t slab_id) {
	hipSetDevice(device_id);
	char *new_slab;
	HANDLE_ERROR(hipMalloc((void **)&new_slab, slab_size));
	slabs[slab_id] = new_slab;
}

char* mog_malloc_gpu(int64_t device_id, 
				int64_t size) {
	hipSetDevice(device_id);
	char *data_p;
	HANDLE_ERROR(hipMalloc((void **)&data_p, size));
	return data_p;
}

void mog_free_gpu(int64_t device_id, 
				char* data_p) {
	hipSetDevice(device_id);
	hipFree ((void*) data_p);
}

void mog_memcpy_cpu_to_gpu(int64_t device_id, char* dst, const char* src, int64_t slabe_size) {
	hipSetDevice(device_id);
	HANDLE_ERROR(hipMemcpy(dst, src, slabe_size, hipMemcpyHostToDevice));
	//cudaDeviceSynchronize();
}

void mog_memcpy_gpu_to_cpu(int64_t device_id, char* dst, const char* src, int64_t slabe_size) {
	hipSetDevice(device_id);
	HANDLE_ERROR(hipMemcpy(dst, src, slabe_size, hipMemcpyDeviceToHost));
	//cudaDeviceSynchronize();
}

void mog_memcpy_gpu_to_gpu(int64_t device_id, char* dst, const char* src, int64_t slabe_size) {
	hipSetDevice(device_id);
	HANDLE_ERROR(hipMemcpy(dst, src, slabe_size, hipMemcpyDeviceToDevice));
	//cudaDeviceSynchronize();
}

/*
void mog_vectorAdd(int64_t device_id, const char *A, const char *B, char *C, int numElements) {
	cudaError_t err = cudaSuccess;
	
	cudaSetDevice(device_id);
	int threadsPerBlock = 256;
    int blocksPerGrid =(1024*1024 + threadsPerBlock - 1) / threadsPerBlock;
    
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
	
	err = cudaGetLastError();
	if (err != cudaSuccess) {
        fprintf(stderr, "Failed to kernel function (error code %s)!\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
*/
