
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void mog_malloc_gpu(int32_t device_id, 
				int32_t page_size, 
				char** pages,
				int32_t page_id) {
	hipSetDevice(device_id);
	char *page;
	HANDLE_ERROR(hipMalloc((void **)&page, page_size));
	pages[page_id] = page;
}

void mog_memcpy_cpu_to_gpu(int32_t device_id, char* dst, const char* src, int32_t page_size) {
	hipSetDevice(device_id);
	HANDLE_ERROR(hipMemcpy(dst, src, page_size, hipMemcpyHostToDevice));
	//cudaDeviceSynchronize();
}

void mog_memcpy_gpu_to_cpu(int32_t device_id, char* dst, const char* src, int32_t page_size) {
	hipSetDevice(device_id);
	HANDLE_ERROR(hipMemcpy(dst, src, page_size, hipMemcpyDeviceToHost));
	//cudaDeviceSynchronize();
}

/*
void mog_vectorAdd(int32_t device_id, const char *A, const char *B, char *C, int numElements) {
	cudaError_t err = cudaSuccess;
	
	cudaSetDevice(device_id);
	int threadsPerBlock = 256;
    int blocksPerGrid =(1024*1024 + threadsPerBlock - 1) / threadsPerBlock;
    
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
	
	err = cudaGetLastError();
	if (err != cudaSuccess) {
        fprintf(stderr, "Failed to kernel function (error code %s)!\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
*/
