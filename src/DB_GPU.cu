
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void db_malloc_gpu(int32_t page_size, int32_t page_num, char** pages) {
	hipSetDevice(0);
	char *page;
	hipMalloc((void **)&page, 1024*page_size*1024);
	pages[0] = page;
	hipSetDevice(1);
	hipMalloc((void **)&page, 1024*page_size*1024);
}
